#include "utils.h"

namespace cu {
int device_count() {
  int count;
  CUDA_CHECK(hipGetDeviceCount(&count));
  return count;
}
};  // namespace cu
