#include "hip/hip_runtime.h"
#include <thrust/complex.h>

#include <cassert>
#include <complex>

#include "gate_operation_impl_cuda.h"
#include "utils.h"

static constexpr size_t BLOCKDIM = 256;

namespace cu {

static __global__ void applyGlobalPhase_kernel(thrust::complex<double> *buffer,
                                               thrust::complex<double> *gate,
                                               size_t target, size_t nqubits,
                                               size_t nelems) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= nelems) return;

  thrust::complex<double> gphase = gate[0];
  buffer[i] = buffer[i] * gphase;
}

static __global__ void applyOneQubitGate_kernel(thrust::complex<double> *buffer,
                                                thrust::complex<double> *gate,
                                                size_t target, size_t nqubits,
                                                size_t nelems) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  size_t st = (1ull << (nqubits - target - 1));

  if (i >= nelems) return;

  if ((i & st) == 0) {
    thrust::complex<double> a0 = buffer[i];
    thrust::complex<double> a1 = buffer[i + st];
    buffer[i] = gate[0 * 2 + 0] * a0 + gate[0 * 2 + 1] * a1;
    buffer[i + st] = gate[1 * 2 + 0] * a0 + gate[1 * 2 + 1] * a1;
  }
}

static __global__ void applyTwoQubitGate_kernel(thrust::complex<double> *buffer,
                                                thrust::complex<double> *gate,
                                                size_t target0, size_t target1,
                                                size_t nqubits, size_t nelems) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  size_t st0 = (1ull << (nqubits - target1 - 1));
  size_t st1 = (1ull << (nqubits - target0 - 1));

  if (i >= nelems) return;

  if ((i & st0) == 0 && (i & st1) == 0) {
    thrust::complex<double> a0 = buffer[i + 0];
    thrust::complex<double> a1 = buffer[i + st0];
    thrust::complex<double> a2 = buffer[i + st1];
    thrust::complex<double> a3 = buffer[i + st1 + st0];
    buffer[i + 0] = gate[0 * 4 + 0] * a0 + gate[0 * 4 + 1] * a1 +
                    gate[0 * 4 + 2] * a2 + gate[0 * 4 + 3] * a3;
    buffer[i + st0] = gate[1 * 4 + 0] * a0 + gate[1 * 4 + 1] * a1 +
                      gate[1 * 4 + 2] * a2 + gate[1 * 4 + 3] * a3;
    buffer[i + st1] = gate[2 * 4 + 0] * a0 + gate[2 * 4 + 1] * a1 +
                      gate[2 * 4 + 2] * a2 + gate[2 * 4 + 3] * a3;
    buffer[i + st1 + st0] = gate[3 * 4 + 0] * a0 + gate[3 * 4 + 1] * a1 +
                            gate[3 * 4 + 2] * a2 + gate[3 * 4 + 3] * a3;
  }
}

static __global__ void applyThreeQubitGate_kernel(
    thrust::complex<double> *buffer, thrust::complex<double> *gate,
    size_t target0, size_t target1, size_t target2, size_t nqubits,
    size_t nelems) {
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  size_t st0 = (1ull << (nqubits - target2 - 1));
  size_t st1 = (1ull << (nqubits - target1 - 1));
  size_t st2 = (1ull << (nqubits - target0 - 1));

  if (i >= nelems) return;

  if ((i & st0) == 0 && (i & st1) == 0 && (i & st2) == 0) {
    thrust::complex<double> a0 = buffer[i + 0];
    thrust::complex<double> a1 = buffer[i + st0];
    thrust::complex<double> a2 = buffer[i + st1];
    thrust::complex<double> a3 = buffer[i + st1 + st0];
    thrust::complex<double> a4 = buffer[i + st2];
    thrust::complex<double> a5 = buffer[i + st2 + st0];
    thrust::complex<double> a6 = buffer[i + st2 + st1];
    thrust::complex<double> a7 = buffer[i + st2 + st1 + st0];
    buffer[i + 0] = gate[0 * 8 + 0] * a0 + gate[0 * 8 + 1] * a1 +
                    gate[0 * 8 + 2] * a2 + gate[0 * 8 + 3] * a3 +
                    gate[0 * 8 + 4] * a4 + gate[0 * 8 + 5] * a5 +
                    gate[0 * 8 + 6] * a6 + gate[0 * 8 + 7] * a7;
    buffer[i + st0] = gate[1 * 8 + 0] * a0 + gate[1 * 8 + 1] * a1 +
                      gate[1 * 8 + 2] * a2 + gate[1 * 8 + 3] * a3 +
                      gate[1 * 8 + 4] * a4 + gate[1 * 8 + 5] * a5 +
                      gate[1 * 8 + 6] * a6 + gate[1 * 8 + 7] * a7;
    buffer[i + st1] = gate[2 * 8 + 0] * a0 + gate[2 * 8 + 1] * a1 +
                      gate[2 * 8 + 2] * a2 + gate[2 * 8 + 3] * a3 +
                      gate[2 * 8 + 4] * a4 + gate[2 * 8 + 5] * a5 +
                      gate[2 * 8 + 6] * a6 + gate[2 * 8 + 7] * a7;
    buffer[i + st1 + st0] = gate[3 * 8 + 0] * a0 + gate[3 * 8 + 1] * a1 +
                            gate[3 * 8 + 2] * a2 + gate[3 * 8 + 3] * a3 +
                            gate[3 * 8 + 4] * a4 + gate[3 * 8 + 5] * a5 +
                            gate[3 * 8 + 6] * a6 + gate[3 * 8 + 7] * a7;
    buffer[i + st2] = gate[4 * 8 + 0] * a0 + gate[4 * 8 + 1] * a1 +
                      gate[4 * 8 + 2] * a2 + gate[4 * 8 + 3] * a3 +
                      gate[4 * 8 + 4] * a4 + gate[4 * 8 + 5] * a5 +
                      gate[4 * 8 + 6] * a6 + gate[4 * 8 + 7] * a7;
    buffer[i + st2 + st0] = gate[5 * 8 + 0] * a0 + gate[5 * 8 + 1] * a1 +
                            gate[5 * 8 + 2] * a2 + gate[5 * 8 + 3] * a3 +
                            gate[5 * 8 + 4] * a4 + gate[5 * 8 + 5] * a5 +
                            gate[5 * 8 + 6] * a6 + gate[5 * 8 + 7] * a7;
    buffer[i + st2 + st1] = gate[6 * 8 + 0] * a0 + gate[6 * 8 + 1] * a1 +
                            gate[6 * 8 + 2] * a2 + gate[6 * 8 + 3] * a3 +
                            gate[6 * 8 + 4] * a4 + gate[6 * 8 + 5] * a5 +
                            gate[6 * 8 + 6] * a6 + gate[6 * 8 + 7] * a7;
    buffer[i + st2 + st1 + st0] = gate[7 * 8 + 0] * a0 + gate[7 * 8 + 1] * a1 +
                                  gate[7 * 8 + 2] * a2 + gate[7 * 8 + 3] * a3 +
                                  gate[7 * 8 + 4] * a4 + gate[7 * 8 + 5] * a5 +
                                  gate[7 * 8 + 6] * a6 + gate[7 * 8 + 7] * a7;
  }
}

static void applyGlobalPhase(thrust::complex<double> *buffer,
                             thrust::complex<double> *gate,
                             std::vector<size_t> targets, size_t nqubits,
                             size_t nelems) {
  applyGlobalPhase_kernel<<<(nelems + BLOCKDIM - 1) / BLOCKDIM, BLOCKDIM>>>(
      buffer, gate, 0, nqubits, nelems);
  CUDA_CHECK(hipGetLastError());
}

static void applyOneQubitGate(thrust::complex<double> *buffer,
                              thrust::complex<double> *gate,
                              std::vector<size_t> targets, size_t nqubits,
                              size_t nelems) {
  applyOneQubitGate_kernel<<<(nelems + BLOCKDIM - 1) / BLOCKDIM, BLOCKDIM>>>(
      buffer, gate, targets[0], nqubits, nelems);
  CUDA_CHECK(hipGetLastError());
}

static void applyTwoQubitGate(thrust::complex<double> *buffer,
                              thrust::complex<double> *gate,
                              std::vector<size_t> targets, size_t nqubits,
                              size_t nelems) {
  applyTwoQubitGate_kernel<<<(nelems + BLOCKDIM - 1) / BLOCKDIM, BLOCKDIM>>>(
      buffer, gate, targets[0], targets[1], nqubits, nelems);
  CUDA_CHECK(hipGetLastError());
}

static void applyThreeQubitGate(thrust::complex<double> *buffer,
                                thrust::complex<double> *gate,
                                std::vector<size_t> targets, size_t nqubits,
                                size_t nelems) {
  applyThreeQubitGate_kernel<<<(nelems + BLOCKDIM - 1) / BLOCKDIM, BLOCKDIM>>>(
      buffer, gate, targets[0], targets[1], targets[2], nqubits, nelems);
  CUDA_CHECK(hipGetLastError());
}

void applyGate(void *_buffer, void *_gate, std::vector<size_t> targets,
               size_t nqubits, size_t nelems) {
  assert(targets.size() == 1 || targets.size() == 2 || targets.size() == 3);
  auto buffer = reinterpret_cast<thrust::complex<double> *>(_buffer);
  auto gate = reinterpret_cast<thrust::complex<double> *>(_gate);
  if (targets.size() == 0) {  // global Phase
    applyGlobalPhase(buffer, gate, targets, nqubits, nelems);
  } else if (targets.size() == 1) {
    applyOneQubitGate(buffer, gate, targets, nqubits, nelems);
  } else if (targets.size() == 2) {
    applyTwoQubitGate(buffer, gate, targets, nqubits, nelems);
  } else if (targets.size() == 3) {
    applyThreeQubitGate(buffer, gate, targets, nqubits, nelems);
  } else {
    assert(false);
  }
}

}  // namespace cu
